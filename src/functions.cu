#include "hip/hip_runtime.h"
/*
 *   DemSo - 2D Discrete Element Method for Soil application
 *   Copyright (C) 2012  UNICAMP FEM/DMC
 *
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "thrust/device_ptr.h"  		   // thrust para utilizar ponteiro
#include "thrust/sort.h" 					   // thrust para ordenar vetor
#include "main.cuh"
#include "particles_kernel.cuh"

// Esse arquivo prepara as funções que serão executadas na GPU. Ele define
// o tamanho do Grid e o número de Threads.

// Aloca espaço na memória da GPU e copia as propriedades para a memória de
// constantes.
void allocateVectors(ParticleProperties* partProps,
					 ParticlesValues* partValues,
					 SystemProperties* sisProps,
					 RenderParameters* renderPar)
{
	// alocando vetores na placa de video
	// hipMalloc --> aloca espaço na placa de vídeo
	// hipMemcpy --> transfere dados entre a CPU (Host) e GPU (Device)
	// hipMemcpyToSymbol --> copia variável para a memória de constante
	hipMalloc((void**)&partValues->type1, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->type2, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->ID1, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->ID2, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->loc1, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->loc2, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->pos1, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->pos2, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->vel1, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->vel2, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->acc, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->theta1, sizeof(float) * sisProps->numParticles);
	hipMalloc((void**)&partValues->theta2, sizeof(float) * sisProps->numParticles);
	hipMalloc((void**)&partValues->omega1, sizeof(float) * sisProps->numParticles);
	hipMalloc((void**)&partValues->omega2, sizeof(float) * sisProps->numParticles);
	hipMalloc((void**)&partValues->alpha, sizeof(float) * sisProps->numParticles);
	hipMalloc((void**)&partValues->cellStart, sizeof(uint) * sisProps->numCells);
	hipMalloc((void**)&partValues->cellEnd, sizeof(uint) * sisProps->numCells);
	hipMalloc((void**)&partValues->gridParticleIndex, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->gridParticleHash, sizeof(uint) * sisProps->numParticles);
	
	// Definindo 0 como valor inicial de todos os vetores alocados acima
	hipMemset(partValues->type1, 0, sizeof(uint) * sisProps->numParticles);
	hipMemset(partValues->type2, 0, sizeof(uint) * sisProps->numParticles);
	hipMemset(partValues->ID1, 0, sizeof(uint) * sisProps->numParticles);
	hipMemset(partValues->ID2, 0, sizeof(uint) * sisProps->numParticles);
	hipMemset(partValues->loc1, 0, sizeof(uint) * sisProps->numParticles);
	hipMemset(partValues->loc2, 0, sizeof(uint) * sisProps->numParticles);
	hipMemset(partValues->pos1, 0, sizeof(float) * sisProps->numParticles * 2);
	hipMemset(partValues->pos2, 0, sizeof(float) * sisProps->numParticles * 2);
	hipMemset(partValues->vel1, 0, sizeof(float) * sisProps->numParticles * 2);
	hipMemset(partValues->vel2, 0, sizeof(float) * sisProps->numParticles * 2);
	hipMemset(partValues->acc, 0, sizeof(float) * sisProps->numParticles * 2);
	hipMemset(partValues->theta1, 0, sizeof(float) * sisProps->numParticles);
	hipMemset(partValues->theta2, 0, sizeof(float) * sisProps->numParticles);
	hipMemset(partValues->omega1, 0, sizeof(float) * sisProps->numParticles);
	hipMemset(partValues->omega2, 0, sizeof(float) * sisProps->numParticles);
	hipMemset(partValues->alpha, 0, sizeof(float) * sisProps->numParticles);
	hipMemset(partValues->cellStart, 0, sizeof(uint) * sisProps->numCells);
	hipMemset(partValues->cellEnd, 0, sizeof(uint) * sisProps->numCells);
	hipMemset(partValues->gridParticleIndex, 0, sizeof(uint) * sisProps->numParticles);
	hipMemset(partValues->gridParticleHash, 0, sizeof(uint) * sisProps->numParticles);


	hipMemcpyToSymbol(HIP_SYMBOL(sisPropD), sisProps, sizeof(SystemProperties));
	hipMemcpyToSymbol(HIP_SYMBOL(renderParD), renderPar, sizeof(RenderParameters));
	hipMemcpyToSymbol(HIP_SYMBOL(partPropD), partProps , sizeof(ParticleProperties) * MAX_PARTICLES_TYPES);
}



// Desaloca o espaço reservado na GPU
void desAllocateVectors(ParticlesValues* partValues)
{
	hipFree( partValues->type1 );
	hipFree( partValues->type2 );
	hipFree( partValues->ID1 );
	hipFree( partValues->ID2 );
	hipFree( partValues->loc1 );
	hipFree( partValues->loc2 );
    hipFree( partValues->pos1 );
    hipFree( partValues->pos2 );
    hipFree( partValues->vel1 );
    hipFree( partValues->vel2 );
    hipFree( partValues->acc );
    hipFree( partValues->theta1 );
    hipFree( partValues->theta2 );
    hipFree( partValues->omega1 );
    hipFree( partValues->omega2 );
    hipFree( partValues->alpha );
    hipFree( partValues->cellStart );
    hipFree( partValues->cellEnd );
    hipFree( partValues->gridParticleIndex );
    hipFree( partValues->gridParticleHash );
}

// Função para retornar o maior inteiro da divisão a/b
inline uint iDivUp(uint a, uint b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

// cria a posição inicial das partículas. Esse kernel é executado em um
// grid 2D com um número máximo de 16 threads por bloco
void createRetangleBlock (float* 		pos,
						  uint*			ID,
						  uint*			loc,
						  uint*			type,
						  float2		start,
						  float2		sideLenght,
						  uint2			side,
						  uint 			startID,
						  uint 			numParticleTypes,
						  uint*			particleTypeVec,
						  unsigned long	seed){

	// alocando vetores na placa de video
	// hipMalloc --> aloca espaço na placa de vídeo
	// hipMemcpy --> transfere dados entre a CPU (Host) e GPU (Device)
	// hipMemcpyToSymbol --> copia variável para a memória de constante
	uint *d_particleTypeVec;

	hipMalloc((void**)&d_particleTypeVec, sizeof(uint)*numParticleTypes);

	hipMemcpy(d_particleTypeVec, particleTypeVec,
			   sizeof(uint)*numParticleTypes, hipMemcpyHostToDevice);

	uint numBlocksx, numBlocksy, numThreadsx, numThreadsy;
	computeGridSize(side.x, 16, numBlocksx, numThreadsx);
	computeGridSize(side.y, 16, numBlocksy, numThreadsy);
	
	dim3 numBlocks(numBlocksx,numBlocksy);
	dim3 numThreads(numThreadsx,numThreadsy);

	createRetangleBlockD<<<numBlocks,numThreads>>>((float2*)pos,
												   ID,
												   loc,
												   type,
												   start,
												   sideLenght,
												   side,
												   startID,
												   numParticleTypes,
												   d_particleTypeVec,
												   seed);
													  
	// Desalocando espaço na placa de vídeo (Não mais necessário)
    hipFree( d_particleTypeVec );													  
}

void createUserDefineBlock (float*	pos,
							float*	vel,
							float*	theta,
							float*	omega,
							uint*	ID,
							uint*	loc,
							uint*	type,
							float2*	usrPos,
							float2* usrVel,
							float*	usrTheta,
							float*	usrOmega,
							uint*	usrType,
							uint	numParticles,
							uint	startID){

	uint IDvec[numParticles];
	for (int i = 0; i < numParticles ; i++){
		IDvec[i] = i + startID;
	}
	
	hipMemcpy(pos,usrPos,sizeof(float)*numParticles*2,hipMemcpyHostToDevice);
	hipMemcpy(vel,usrVel,sizeof(float)*numParticles*2,hipMemcpyHostToDevice);
	hipMemcpy(theta,usrTheta,sizeof(float)*numParticles,hipMemcpyHostToDevice);
	hipMemcpy(omega,usrOmega,sizeof(float)*numParticles,hipMemcpyHostToDevice);
	hipMemcpy(type,usrType,sizeof(uint)*numParticles,hipMemcpyHostToDevice);
	hipMemcpy(ID,IDvec,sizeof(uint)*numParticles,hipMemcpyHostToDevice);
	hipMemcpy(loc,IDvec,sizeof(uint)*numParticles,hipMemcpyHostToDevice);
							
}

// Calcula o numero da celula de cada particula. Esse kernel é executado
// em um grid 1D com um número máximo de 256 threads por bloco
void calcHash(float* 	pos,
			  uint* 	gridParticleIndex,
			  uint* 	gridParticleHash,
			  uint 		numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float2*)pos);
}

// Ordena as partículas com base no número do Hash. Essa rotina é executada
// pela biblioteca THRUST.
// A função device_ptr permite passar o ponteiro de uma variável alocada na
// GPU para o thrust.
// Em seguida a função sort_by_key organiza o vetor dGridParticleHash em
// ordem crescente e arruma o vetor dGridParticleIndex com base na
// ordenação
void sortParticles(uint* dGridParticleHash, uint* dGridParticleIndex, uint numParticles)
{
    thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                        thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                        thrust::device_ptr<uint>(dGridParticleIndex));
}

// Reordena os vetores de posição e velocidade com base na nova ordem das
// partículas. Em seguida o vetor de inicio e fim de cada célula é criado.
// Esse kernel é executado em um grid 1D com um número máximo de 256
// threads por bloco
void reorderDataAndFindCellStart(uint*  cellStart,
							     uint*  cellEnd,
							     float* sortedPos,
							     float* sortedVel,
								 float* sortedTheta,
								 float* sortedOmega,
							     uint* 	sortedID,
							     uint* 	sortedLoc,
							     uint* 	sortedType,
                                 uint*  gridParticleHash,
                                 uint*  gridParticleIndex,
							     float* oldPos,
							     float* oldVel,
								 float* oldTheta,
								 float* oldOmega,
							     uint*	oldID,
							     uint* 	oldType,
							     uint   numParticles,
							     uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // set all cells to empty
	hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

	// Declarando como memória de textura
	#if USE_TEX
		hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float2));
		hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float2));
		hipBindTexture(0, oldIDTex, oldID, numParticles*sizeof(uint));
		hipBindTexture(0, oldTypeTex, oldType, numParticles*sizeof(uint));
	#endif

    uint smemSize = sizeof(uint)*(numThreads+1);
    reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
        cellStart,
        cellEnd,
        (float2*)sortedPos,
        (float2*)sortedVel,
		sortedTheta,
		sortedOmega,
        sortedID,
        sortedLoc,
        sortedType,
		gridParticleHash,
		gridParticleIndex,
        (float2*)oldPos,
        (float2*)oldVel,
		oldTheta,
		oldOmega,
        oldID,
        oldType);
    
    // Retirando da memória de textura 
	#if USE_TEX
		hipUnbindTexture(oldPosTex);
		hipUnbindTexture(oldVelTex);
		hipUnbindTexture(oldIDTex);
		hipUnbindTexture(oldTypeTex);
	#endif

}

// Rotina que verifica a colisão entre as partículas e transforma a força
// de colisão em aceleração. Esse kernel é executado em um grid 1D com um
// número máximo de 64 threads por bloco
void collide(float* 	oldPos,
             float* 	oldVel,
             float* 	newAcc,
			 float*		oldOmega,
			 float*		newAlpha,
             uint*		oldType,
             uint*  	cellStart,
             uint*  	cellEnd,
             uint   	numParticles,
             uint 		numCells
#if USE_BIG_PARTICLE
			 , float2		controlPos,
			 uint		controlType
#endif
			 )
{
	// Declarando como memória de textura
	#if USE_TEX
		hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float2));
		hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float2));
		hipBindTexture(0, oldTypeTex, oldType, numParticles*sizeof(uint));
		hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint));
		hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint));    
	#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    collideD<<< numBlocks, numThreads >>>((float2*)oldPos,
                                          (float2*)oldVel,
                                          (float2*)newAcc,
										  oldOmega,
										  newAlpha,
                                          oldType,
                                          cellStart,
                                          cellEnd
#if USE_BIG_PARTICLE
			 							  , controlPos,
										  controlType
#endif
										  );
										  
    // Retirando da memória de textura 
	#if USE_TEX
		hipUnbindTexture(oldPosTex);
		hipUnbindTexture(oldVelTex);
		hipUnbindTexture(oldTypeTex);
		hipUnbindTexture(cellStartTex);
		hipUnbindTexture(cellEndTex);
	#endif
}

// Realiza a integração numérica do sistema. Essa é uma integração linear,
// onde:
// Velocidade = Velocidade + Aceleração * DeltaTempo
// Posicão    =  Posição   + Velocidade * DeltaTempo
// Esse kernel é executado em um grid 1D com um número máximo de 256
// threads por bloco.
void integrateSystem(float*	pos,
					 float*	vel,
					 float*	acc,
					 float* theta,
					 float* omega,
					 float* alpha,
					 uint*	type,
					 uint	numParticles)
{
	uint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	
	// execute the kernel
	integrateSystemD<<<numBlocks,numThreads>>>((float2*)pos,
				 							   (float2*)vel,
				 							   (float2*)acc,
											   theta,
											   omega,
											   alpha,
				 							   type);
}

// Desenha as partículas em uma imagem de DIMx x DIMy pixels e mostra na
// tela. O fundo da imagem é definido como preto e as partículas são
// brancas. Esse kernel é executado em um grid 1D com um número máximo de
// 256 threads por bloco.
void plotParticles(uchar4*	ptr,
				   float* 	pos,
				   float*	theta,
				   uint*	type,
				   uint 	numParticles,
				   int 		DIMx,
				   int		DIMy
#if USE_BIG_PARTICLE
				   ,float2 	controlPos,
				   uint		controlType,
				   int		dimx,
				   int		dimy
#endif
				   ){

	// pinta o fundo de preto
	hipMemset(ptr, 0, DIMx*DIMy*sizeof(uchar4));
	
	uint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	
	// execute the kernel
	plotSpheresD<<<numBlocks,numThreads>>>(ptr,
									 	   (float2*)pos,
										   theta,
									 	   type);

#if USE_BIG_PARTICLE
	uint numBlocksx, numBlocksy, numThreadsx, numThreadsy;
	computeGridSize(dimx, 16, numBlocksx, numThreadsx);
	computeGridSize(dimy, 16, numBlocksy, numThreadsy);
	
	dim3 numBlocks2(numBlocksx,numBlocksy);
	dim3 numThreads2(numThreadsx,numThreadsy);

	// execute the kernel
	plotControlParticleD<<<numBlocks2,numThreads2>>>(ptr,
													 controlPos,
													 controlType);
#endif
}

// Escreve no arquivo de saída os dados desejados.
// O arquivo de saída é do tipo texto. Na primeira linha encontra-se
// o valor do timeStep. Em seguida, cada linha apresenta, separados
// por vírgulas, o número da iteracão, e cada um dos dados de saída
// desejados.
// TODO:
//  - Seguir uma dada partícula (por enquanto ele se perde no sort)
//  - Checar: Aparecimento de varios NaN quando shearStiffness = 1000
void writeOutputFile (DataBlock *simBlock, int ticks)
{
	// Shortcuts
    ParticlesValues *partValues = &simBlock->partValues;
	TimeControl *timeCtrl = &simBlock->timeCtrl;
	FILE * outputFile = simBlock->outputFile;

	// Chosen particle's index
//	const int blah = 10;
//	for (register int i = 0; i < simBlock->sisProps.numParticles; i++)
//	{
//		if (partValues[i].ID)
//	}

	int chosenOne = 10;

	// Copying data from the GPU
	// Iteration number
	int h_iteration;
	h_iteration = timeCtrl->tempo;
	//hipMemcpy (&h_iteration, &timeCtrl->tempo, sizeof(int), hipMemcpyDeviceToHost);
	
	// Particle Data
	float h_pos[2], h_vel[2], h_acc[2];
	float h_theta, h_omega, h_alpha;
	uint h_id, h_type;

	// Geting the right particle data
	if (ticks & 1) // quando par (FALSE) quando impar (TRUE)
	{	
		hipMemcpy (&h_pos,   &partValues->pos2[chosenOne],   2*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy (&h_vel,   &partValues->vel2[chosenOne],   2*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy (&h_theta, &partValues->theta2[chosenOne], sizeof(float),   hipMemcpyDeviceToHost);
		hipMemcpy (&h_omega, &partValues->omega2[chosenOne], sizeof(float),   hipMemcpyDeviceToHost);
		hipMemcpy (&h_id,	  &partValues->ID2[chosenOne],    sizeof(uint),    hipMemcpyDeviceToHost);
		hipMemcpy (&h_type,  &partValues->type2[chosenOne],  sizeof(uint),    hipMemcpyDeviceToHost);
	} else {
		hipMemcpy (&h_pos,   &partValues->pos1[chosenOne],   2*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy (&h_vel,   &partValues->vel1[chosenOne],   2*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy (&h_theta, &partValues->theta1[chosenOne], sizeof(float),   hipMemcpyDeviceToHost);
		hipMemcpy (&h_omega, &partValues->omega1[chosenOne], sizeof(float),   hipMemcpyDeviceToHost);
		hipMemcpy (&h_id,	  &partValues->ID1[chosenOne],    sizeof(uint),    hipMemcpyDeviceToHost);
		hipMemcpy (&h_type,  &partValues->type1[chosenOne],  sizeof(uint),	   hipMemcpyDeviceToHost);
	}
	hipMemcpy (&h_acc,   &partValues->acc[chosenOne],   2*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy (&h_alpha, &partValues->alpha[chosenOne], sizeof(float),   hipMemcpyDeviceToHost);

	// Printing to file
	// Iteration number
	fprintf (outputFile, "%d,", h_iteration); // Don't print newline

	// Particle Data
	fprintf (outputFile, "%u,%u,%f,%f,%f,%f,%f,%f,%f,%f,%f\n", h_id, h_type,
			 h_pos[0], h_pos[1], h_vel[0], h_vel[1], h_acc[0], h_acc[1],
			 h_theta, h_omega, h_alpha);
}
