#include "hip/hip_runtime.h"

// Includes que eu acho que preciso
//#include <algorithm>
//#include <assert.h>
//#include <cstdio>
//#include <cstdlib>
#include <math.h>
//#include <memory.h>
#include <stdio.h>
//#include <stdlib.h>
//#include <string.h>
#include <iostream>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../includes/cuda_by_example.h"
#include "../includes/gpu_anim.h"                   // este inclui o hip/hip_runtime.h
#include "../includes/cutil_math.h"

// THRUST includes
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"


// Headers


// Dependece files
#include "particles_kernel.cu"
#include "functions.cu"

#define DIM 800
#define PARTICLES 20000
#define BOX_SIZE 10.0f
#define TIME_STEP 1.0e-3
#define GRAVITY 9.81f
#define BOUNDARYDAMPING -0.5f
#define X_PARTICLES 150
#define Y_PARTICLES 140

#define log2( x ) log(x)/log(2)

void PrepareSim( SistemProperties *params, ParticlesValues *particle, ParticleProperties *partProps ){
	
//	partProps = (ParticleProps*)malloc( sizeof(ParticleProps) * 1);

	params->numParticles = PARTICLES;

	params->cubeDimension.x = params->cubeDimension.y = BOX_SIZE;
	
	params->timeStep = TIME_STEP;
	
	params->gravity = make_float2(0,-GRAVITY);
		
	partProps[0].radius = 20e-3f;
	partProps[0].mass = 1e-2;
	partProps[0].collideStiffness = 1e3;
	partProps[0].collideDamping = 0.1f;
	partProps[0].boundaryDamping = BOUNDARYDAMPING;

	// Bloco inicial de esferas
	float corner1[2] = {0.1, 0.1};
	float corner2[2] = {9.9, 9.9};
	float sideLenght[2];

	// Grid dimension
	uint grid = params->cubeDimension.x / (4.0f * partProps[0].radius);
	uint temp = log2(grid);
	uint gridUpdate = pow(2,temp);
	float cellSize = params->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * partProps[0].radius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * partProps[0].radius ) temp += 1;
	params->gridSize.x = pow(2,temp);
	
	grid = params->cubeDimension.y / (4 * partProps[0].radius);
	temp = log2(grid);
	gridUpdate = pow(2,temp);
	cellSize = params->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * partProps[0].radius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * partProps[0].radius ) temp += 1;	
	params->gridSize.y = pow(2,temp);

	params->numCells = params->gridSize.x * params->gridSize.y;

	// Posicionando as primeiras particulas
	sideLenght[0] = corner2[0] - corner1[0];
	sideLenght[1] = corner2[1] - corner1[1];
	
	uint side[2] = {X_PARTICLES, Y_PARTICLES};
	
	// alocando vetores na placa de video
	float *d_corner1, *d_sideLenght;
	uint *d_side;

	hipMalloc((void**)&d_corner1, sizeof(float)*2);
	hipMalloc((void**)&d_sideLenght, sizeof(float)*2);
	hipMalloc((void**)&d_side, sizeof(uint)*2);
	hipMalloc((void**)&particle->pos1, sizeof(float2) * params->numParticles);
	hipMalloc((void**)&particle->pos2, sizeof(float2) * params->numParticles);
	hipMalloc((void**)&particle->vel1, sizeof(float2) * params->numParticles);
	hipMalloc((void**)&particle->vel2, sizeof(float2) * params->numParticles);
	hipMalloc((void**)&particle->acc, sizeof(float2) * params->numParticles);
	hipMalloc((void**)&particle->cellStart, sizeof(uint) * params->numCells);
	hipMalloc((void**)&particle->cellEnd, sizeof(uint) * params->numCells);
	hipMalloc((void**)&particle->particleIndex, sizeof(uint) * params->numParticles);
	hipMalloc((void**)&particle->particleHash, sizeof(uint) * params->numParticles);
	hipMemcpy(d_corner1, corner1, sizeof(float)*2, hipMemcpyHostToDevice);
	hipMemcpy(d_sideLenght, sideLenght, sizeof(float)*2, hipMemcpyHostToDevice);
	hipMemcpy(d_side, side, sizeof(uint)*2, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(simPropD), params, sizeof(SistemProperties));
	hipMemcpyToSymbol(HIP_SYMBOL(partPropD), partProps , sizeof(ParticleProperties) * 1);

	initializeParticlePosition(particle->pos1,
							   particle->vel1,
							   particle->acc,
							   d_corner1,
							   d_sideLenght,
							   d_side,
							   side);

    hipFree( d_corner1 );
    hipFree( d_sideLenght );
    hipFree( d_side );


	// Screen output	
	printf("Number of Spheres = %d\n",params->numParticles);
	printf("grid %d x %d\n",params->gridSize.x,params->gridSize.y);
}

void SimLooping( uchar4 *pixels, DataBlock *simBlock, int ticks ) {

    SistemProperties *sisProps = &simBlock->sisProps;
    ParticleProperties *partProps = &simBlock->partProps;
    ParticlesValues *partValues = &simBlock->partValues;

	float2 *oldPos, *oldVel, *sortPos, *sortVel;

	if ((ticks % 2))
	{	
//		printf("1");
		oldPos = partValues->pos1;
		sortPos = partValues->pos2;
		oldVel = partValues->vel1;
		sortVel = partValues->vel2;
	} else {
//		printf("0");
		oldPos = partValues->pos2;
		sortPos = partValues->pos1;
		oldVel = partValues->vel2;
		sortVel = partValues->vel1;
	}
	
		// Define a celula de cada particula
		calcHash(oldPos,
				 partValues->particleIndex,
				 partValues->particleHash,
				 sisProps->numParticles);

		// Ordena o grid pela posicao das particulas
		sortParticles(partValues->particleHash,
					  partValues->particleIndex,
					  sisProps->numParticles);

		// Encontra as particulas de inicializacao e de finalizacao
		reorderDataAndFindCellStart(partValues->cellStart,
									partValues->cellEnd,
									sortPos,
									sortVel,
									partValues->particleHash,
									partValues->particleIndex,
									oldPos,
									oldVel,
									sisProps->numParticles,
									sisProps->numCells);

		// Detecta a colizao das particulas
		collide(sortPos,
				sortVel,
				partValues->acc,
				partValues->particleIndex,
				partValues->cellStart,
				partValues->cellEnd,
				sisProps->numParticles,
				sisProps->numCells);

//		// Integracao no tempo (atualizacao das posicoes e velocidades)
		integrateSystem(sortPos,
			 	  		sortVel,
			 	  		partValues->acc,
			 	  		sisProps->numParticles);

		// Saida grarica quando necessario
		plotParticles(pixels,
					  sortPos,
					  sisProps->numParticles,
					  sisProps->cubeDimension,
					  partProps->radius,
					  DIM);

//printf("Fim %d\n\n",ticks);

}

void FinalizingSim( DataBlock *simBlock) {

    // Limpe aqui o que tiver que ser limpo
    
    SistemProperties *d1 = &simBlock->sisProps;
    ParticleProperties *d3 = &simBlock->partProps;
    ParticlesValues *d2 = &simBlock->partValues;
    
    hipFree( d2->pos1 );
    hipFree( d2->pos2 );
    hipFree( d2->vel1 );
    hipFree( d2->vel2 );
    hipFree( d2->acc );
    hipFree( d2->cellStart );
    hipFree( d2->cellEnd );
    hipFree( d2->particleIndex );
    hipFree( d2->particleHash );
    hipFree( d1 );
    hipFree( d2 );
    hipFree( d3 );

}


int main() {

    DataBlock simBlock;
    
    SistemProperties *sisProps = &simBlock.sisProps;
    ParticleProperties *partProps = &simBlock.partProps;
    ParticlesValues *partValues = &simBlock.partValues;
    
    GPUAnimBitmap bitmap(DIM, DIM, &simBlock );

	// Utilizar ARGC e ARGV para pegar propriedades na linha de comando
	// ler esses comandos de um arquivo TXT externo
	// Criar uma rotina para fazer este tipo de leitura
	
	// Prepara a simulacao, define as condicoes iniciais do problema
	PrepareSim(sisProps, partValues, partProps);

    bitmap.anim_and_exit(
        (void (*)(uchar4*,void*,int))SimLooping, (void (*)(void*))FinalizingSim );

}
