#include "hip/hip_runtime.h"
/*
 *   DemSo - 2D Discrete Element Method for Soil application
 *   Copyright (C) 2012  UNICAMP FEM/DMC
 *
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

// Input and Output include
#include <iostream>								  // entra e saída de dados
#include <stdio.h>
#include <ctime> 		   // biblioteca de tempo para criar o seed do rand

// CUDA includes
#include "gpu_anim.h" 				  // bib. de vizualização em tempo real
#include "cutil_math.h" 		      // funções matemáticas de vetores

// Dependece files
#include "main.cuh"
#include "functions.cuh" 	 // arquivo de funções de preparação para a GPU
#include "datatypes.hpp"
#include "parser.hpp"


#define log2( x ) log(x)/log(2)

using std::cout;
using std::endl;

bool file_exist (const char *filename)
{
	if (FILE *file = fopen(filename, "r"))
	{
		fclose(file);
		return true;
	} else {
		return false;
	}
}

void PrepareSim (const char *filename,
				 DataBlock *simBlock,
				 ParticleProperties *partProps)
{
	// Estruturas auxiliares
    SystemProperties *sisProps = &simBlock->sisProps;
    ParticlesValues *partValues = &simBlock->partValues;
	RenderParameters *renderPar = &simBlock->renderPar;

	/* Usamos a estrutura de dados C++ e carregamos o arquivo de estado */
	DEMSimulation sim;
	sim.loadFromFile(filename);
	sim.printConfiguration();
	/* Agora vamos copiar para a estrutura C */

	// Número de partículas no sistema é o número de partículas do bloco
	// mais o número de partículas avulsas
	sisProps->numParticles = sim.particles.num.x * sim.particles.num.y
							 + sim.particles.pos.size();

	sisProps->cubeDimension = sim.environment.dimension;

	sisProps->timeStep = sim.parameters.timeStep;

	simBlock->followedParticles = sim.parameters.followedParticles;
	
	sisProps->gravity = sim.environment.gravity;

	sisProps->boundaryNormalStiffness = sim.environment.boundaryNormalStiffness;
	sisProps->boundaryShearStiffness = sim.environment.boundaryShearStiffness;
	
	renderPar->imageDIMy = sim.parameters.imageDIMy;
	renderPar->imageDIMx = sisProps->cubeDimension.x/sisProps->cubeDimension.y*renderPar->imageDIMy;

	// PARSER: copiando as propriedades de partículas
	for (register int i = 0; i < sim.properties.particleTypes.size(); i++)
	{
		partProps[i].mass = sim.properties.particleTypes[i].mass;
		partProps[i].radius = sim.properties.particleTypes[i].radius;
		partProps[i].normalStiffness = sim.properties.particleTypes[i].normalStiffness;
		partProps[i].shearStiffness = sim.properties.particleTypes[i].shearStiffness;
		partProps[i].normalDamping = sim.properties.particleTypes[i].normalDamping;
		partProps[i].boundaryDamping = sim.properties.particleTypes[i].boundaryDamping;
		partProps[i].frictionCoefficient = sim.properties.particleTypes[i].frictionCoefficient;
		partProps[i].colorR = sim.properties.particleTypes[i].color.x;
		partProps[i].colorG = sim.properties.particleTypes[i].color.y;
		partProps[i].colorB = sim.properties.particleTypes[i].color.z;

		partProps[i].inertia = partProps[i].mass*partProps[i].radius*partProps[i].radius / 2;
	}

	// Definindo o maior raio da simulação
	// Se existir uma única partícula gigante (TUBULAÇÃO) talvez seja
	// interessante desprezar esse valor e no lugar, fazer com que essa
	// única partícula teste com todas as outras.
	float maxRadius = 0, plotRadius;
	for (int i = 0; i < sim.properties.particleTypes.size()
#if USE_BIG_PARTICLE
	-1
#endif
	; i++){
		if (maxRadius < partProps[i].radius) maxRadius = partProps[i].radius;
	}

#if USE_BIG_PARTICLE
	if (maxRadius < partProps[sim.properties.particleTypes.size()-1].radius){
		plotRadius = partProps[sim.properties.particleTypes.size()-1].radius;
	}else{
#endif
		plotRadius = maxRadius;
#if USE_BIG_PARTICLE
	}
#endif
	// tamanho do quadrado que contem a esfera em PIXEL (para a saida grafica)
	renderPar->dimx = ceil(renderPar->imageDIMx/sisProps->cubeDimension.x*plotRadius)*2;
	if (renderPar->dimx < 2) renderPar->dimx = 2;
	renderPar->dimy = ceil(renderPar->imageDIMy/sisProps->cubeDimension.y*plotRadius)*2;
	if (renderPar->dimy < 2) renderPar->dimy = 2;
	
	// raio da esfera em PIXEL (para a saída grafica)

//	renderPar->pRadius = renderPar->imageDIMy/sisProps->cubeDimension.y*maxRadius;


	float start[2];
	start[0] = sim.particles.start.x;
	start[1] = sim.particles.start.y;

	// Calcula o tamanho do grid arredondando para um valor que seja
	// potencia de 2. O grid deve ser de 1.2 a 3 vezes o diametro da esfera
	uint grid = sisProps->cubeDimension.x / (4.0f * maxRadius);
	uint temp = log2(grid);
	uint gridUpdate = 1 << temp;
	float cellSize = sisProps->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * maxRadius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * maxRadius ) temp += 1;
	sisProps->gridSize.x = 1 << temp;
	
	grid = sisProps->cubeDimension.y / (4 * maxRadius);
	temp = log2(grid);
	gridUpdate = 1 << temp;
	cellSize = sisProps->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * maxRadius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * maxRadius ) temp += 1;	
	sisProps->gridSize.y = 1 << temp;

	sisProps->numCells = sisProps->gridSize.x * sisProps->gridSize.y;
	
	// Bloco inicial de esferas
	float sideLenght[2];
	sideLenght[0] = sim.particles.end.x - sim.particles.start.x; 			   // dimensao em X
	sideLenght[1] = sim.particles.end.y - sim.particles.start.y; 			   // dimensao em Y
	
	uint side[2];
	side[0] = sim.particles.num.x;
	side[1] = sim.particles.num.y;
	
	allocateVectors(partProps, partValues, sisProps, renderPar);

	// Função para definir a posição inicial das esferas
	initializeParticlePosition(partValues->pos1,
							   partValues->vel1,
							   partValues->acc,
							   partValues->theta1,
							   partValues->omega1,
							   partValues->alpha,
							   partValues->ID1,
							   partValues->loc1,
							   partValues->type1,
							   start,
							   sideLenght,
							   side,
							   time(NULL),
#if USE_BIG_PARTICLE
							   sim.properties.particleTypes.size()-1); // subraindo 1 para não fazer o sorteio com a partícula controlada ********************************************
#else
							   sim.properties.particleTypes.size());
#endif

#if USE_BIG_PARTICLE
	float2 bigParticlePos = make_float2(5,1.5);

//	// Adicionar partícula externa gigante
//	initializeBigParticlePosition(partValues->controlPos,
//								  bigParticlePos);
								  
	partValues->controlPos = bigParticlePos;
	partValues->controlType = sim.properties.particleTypes.size()-1; // Tipo da partícula, por enquanto ela é a última **************************************************************
#endif

	// Screen output	
	printf("\nNumero de Particulas = %d\n", sisProps->numParticles);
	printf("grid %d x %d\n\n", sisProps->gridSize.x, sisProps->gridSize.y);

#if USE_TEX
	printf("Memoria de textura: UTILIZADA\n\n");
#else
	printf("Memoria de textura: NAO\n\n");
#endif 
}

void SimLooping( uchar4 *image, DataBlock *simBlock, int ticks ) {

	// Estruturas auxiliares
    SystemProperties *sisProps = &simBlock->sisProps;
    ParticlesValues *partValues = &simBlock->partValues;
	RenderParameters *renderPar = &simBlock->renderPar;
	TimeControl *timeCtrl = &simBlock->timeCtrl;

	// inicia o cronometro
	timeCtrl->start = clock();
	
	// para ordenarmos os vetores de posicao e velocidade sem necessidade
	// de retornarmos a variável para o vetor original, um switch entre os
	// dois vetores de posição alocados na GPU é criado. A cada iteração o
	// vetor de início e o vetor reorganizado são invertidos, reduzindo uma
	// operação de cópia
	float  *oldPos,  *oldVel;
	float *sortPos, *sortVel;
	float *oldTheta, *oldOmega;
	float *sortTheta, *sortOmega;
	uint  *oldID,  *oldType;
	uint *sortID, *sortType, *sortLoc;

	
	// Integrando o programa IPS vezes antes de exibir a imagem
	for (int i = 0 ; i < timeCtrl->IPS ; i++) {

		if ((ticks + i) & 1) // quando par (FALSE) quando impar (TRUE)
		{	
			oldPos = partValues->pos1;
			oldVel = partValues->vel1;
			oldTheta = partValues->theta1;
			oldOmega = partValues->omega1;
			oldID = partValues->ID1;
			oldType = partValues->type1;
			sortPos = partValues->pos2;
			sortVel = partValues->vel2;
			sortTheta = partValues->theta2;
			sortOmega = partValues->omega2;
			sortID = partValues->ID2;
			sortLoc = partValues->loc2;
			sortType = partValues->type2;
		} else {
			oldPos = partValues->pos2;
			oldVel = partValues->vel2;
			oldTheta = partValues->theta2;
			oldOmega = partValues->omega2;
			oldID = partValues->ID2;
			oldType = partValues->type2;
			sortPos = partValues->pos1;
			sortVel = partValues->vel1;
			sortTheta = partValues->theta1;
			sortOmega = partValues->omega1;
			sortID = partValues->ID1;
			sortLoc = partValues->loc1;
			sortType = partValues->type1;
		}
		
//		// Integracao no tempo (atualizacao das posicoes e velocidades)
//		integrateSystem(oldPos,
//			 	  		oldVel,
//			 	  		partValues->acc,
//						oldTheta,
//						oldOmega,
//						partValues->alpha,
//			 	  		oldType,
//			 	  		sisProps->numParticles);

		// Define a celula de cada particula, criando os vetores
		// gridParticleIndex e gridParticleHash ordenados pelo Index
		calcHash(oldPos,
				 partValues->gridParticleIndex,
				 partValues->gridParticleHash,
				 sisProps->numParticles);

		// Reordena os vetores baseado no Hash
		sortParticles(partValues->gridParticleHash,
					  partValues->gridParticleIndex,
					  sisProps->numParticles);

		// Reorganiza as variaveis de Pos e Vel para a nova ordem de particulas
		// e cria os vetores indicando a partícula de início e fim de cada
		// celula
		reorderDataAndFindCellStart(partValues->cellStart,
									partValues->cellEnd,
									sortPos,
									sortVel,
									sortTheta,
									sortOmega,
									sortID,
									sortLoc,
									sortType,
									partValues->gridParticleHash,
									partValues->gridParticleIndex,
									oldPos,
									oldVel,
									oldTheta,
									oldOmega,
									oldID,
									oldType,
									sisProps->numParticles,
									sisProps->numCells);

		// Detecta a colisao das particulas e transforma a força de colisão em
		// aceleração
		collide(sortPos,
				sortVel,
				partValues->acc,
				sortOmega,
				partValues->alpha,
				sortType,
				partValues->cellStart,
				partValues->cellEnd,
				sisProps->numParticles,
				sisProps->numCells
#if USE_BIG_PARTICLE
				, partValues->controlPos,
				partValues->controlType
#endif
				);

		// Integracao no tempo (atualizacao das posicoes e velocidades)
		integrateSystem(sortPos,
			 	  		sortVel,
			 	  		partValues->acc,
						sortTheta,
						sortOmega,
						partValues->alpha,
			 	  		sortType,
			 	  		sisProps->numParticles);


#if USE_BIG_PARTICLE
		partValues->controlPos.y += -.005;
		partValues->controlPos.x += .000;
		if (partValues->controlPos.y < -1) partValues->controlPos.y = 9.5;
		if (partValues->controlPos.x > sisProps->cubeDimension.x + 25.5) partValues->controlPos.x = -25.5;
#endif

		timeCtrl->tempo++;
	}

	// Saida grarica quando necessario
	plotParticles(image,
				  sortPos,
				  sortTheta,
				  sortType,
				  sisProps->numParticles,
				  renderPar->imageDIMx,
				  renderPar->imageDIMy
#if USE_BIG_PARTICLE
				  , partValues->controlPos,
				  partValues->controlType,
				  renderPar->dimx,
				  renderPar->dimy
#endif
				  );

	// Escreve no arquivo de output os dados de saída
	if (!simBlock->followedParticles.empty())
		writeOutputFile (simBlock->outputFile,
						 simBlock->followedParticles,
						 sisProps->timeStep * timeCtrl->tempo, // Current elapsed time
						 (float2*)sortPos,
						 (float2*)sortVel,
						 (float2*)partValues->acc,
						 sortTheta,
						 sortOmega,
						 partValues->alpha,
						 sortID,
						 sortType,
						 sortLoc);
	
	// calcula o tempo de exibição do frame
	double time = ((double)clock() - timeCtrl->start)/CLOCKS_PER_SEC;
	if (time < 0.003f) time = 0.03f;
	
	// Define o número de Interações por segundo para exibir a imagem em 
	// FPS (definida no cabeçalho) frames por segundo.
	// Após a conta, transforma o número em impar para não calcular duas
	// duas vezes a mesma iteração (por causa do switch)
	timeCtrl->IPS = floor(1.0f/time/FPS*timeCtrl->IPS);
	timeCtrl->IPS = timeCtrl->IPS | 0x0001;

}

void FinalizingSim( DataBlock *simBlock) {

	TimeControl *timeCtrl = &simBlock->timeCtrl;

    // Limpe aqui o que tiver que ser limpo
	desAllocateVectors( &simBlock->partValues );
    
   	printf("Integracoes por plot = %d\n\n",timeCtrl->IPS);
	double elapsedTime = ((double)clock() - timeCtrl->totalStart)/CLOCKS_PER_SEC;
	double simulationTime = timeCtrl->tempo * simBlock->sisProps.timeStep;
	
	printf("Duracao da simulacao = %4.2f s\n",elapsedTime);
	printf("Tempo de simulacao = %4.2f s\n\n",simulationTime);
	printf("Razao de tempo (Real/Simulado) = %3.3f\n\n",elapsedTime/simulationTime);
	
}


int main(int argc, char **argv)
{	
	// Verificando arquivo de entrada (Parametros da simulacao)
	char *filename;
	
	if (argc == 2){
		if (file_exist(argv[1])){
			printf("\nUsing %s parameters file\n\n",argv[1]);
			filename = argv[1];
		}else if (file_exist("exemplos/default.dsml")){
			printf("\nFile %s does not exist, using exemplos/default.dsml file\n\n",argv[1]);
			filename = "exemplos/default.dsml";
		}else{
			printf("\nFile %s and exemlos/default.dsml does not exist.\n\nClosing simulation\n\n",argv[1]);
			return 0;
		}
	}else if (argc == 1){
		if (file_exist("exemplos/default.dsml")){
			printf("\nUsing default parameters file (exemplos/default.dsml)\n\n");
			filename = "exemplos/default.dsml";
		}else{
			printf("\nDefault file exemlos/default.dsml does not exist.\n\nClosing simulation\n\n");
			return 0;
		}
	}else{
		printf("\nToo many arguments.\n\nClosing simulation\n\n");
		return 0;
	}
	
	// declarando estrutura de dados principal
    DataBlock simBlock;
    
    // declarando as subestruturas (apenas por facilidade)
    //SystemProperties *sisProps = &simBlock.sisProps;
    //ParticlesValues *partValues = &simBlock.partValues;
	RenderParameters *renderPar = &simBlock.renderPar;
	TimeControl *timeCtrl = &simBlock.timeCtrl;
    
    ParticleProperties partProps[MAX_PARTICLES_TYPES];
    
    // Definindo que a primeira iteração será exibida
    timeCtrl->IPS = 1;
	timeCtrl->totalStart = clock();
	timeCtrl->tempo = 0;
	
	// Prepara a simulacao, define as condicoes iniciais do problema
	PrepareSim(filename, &simBlock, partProps);
	

    // função que define o tamanho da imagem e a estrutura que será
    // repassada para dentro do looping
    GPUAnimBitmap bitmap(renderPar->imageDIMx, renderPar->imageDIMy, &simBlock );
	

	// Abre arquivo de output
	simBlock.outputFile = fopen ("output.csv", "w");

	// Executa o looping até que a tecla ESC seja pressionada
    bitmap.anim_and_exit(
        (void (*)(uchar4*,void*,int))SimLooping, (void (*)(void*))FinalizingSim );

	// Fecha arquivo de output
	fclose (simBlock.outputFile);
	
	return 0;
}
