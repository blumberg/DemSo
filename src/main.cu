#include "hip/hip_runtime.h"
// Input and Output include
#include <iostream>								  // entra e saída de dados
#include <time.h> 		   // biblioteca de tempo para criar o seed do rand

// CUDA includes
#include "gpu_anim.h" 				  // bib. de vizualização em tempo real
#include "cutil_math.h" 		      // funções matemáticas de vetores

// Dependece files
#include "main.cuh"
#include "functions.cuh" 	 // arquivo de funções de preparação para a GPU


#define log2( x ) log(x)/log(2)

void PrepareSim( SystemProperties *sisProps,
				 ParticlesValues *partValues,
				 ParticleProperties *partProps ) {

	sisProps->numParticles = PARTICLES;

	sisProps->cubeDimension.x = sisProps->cubeDimension.y = BOX_SIZE;
	
	sisProps->timeStep = TIME_STEP;
	
	sisProps->gravity = make_float2(0,-GRAVITY);
	
	sisProps->imageDIMx = DIM;
	sisProps->imageDIMy = DIM;
		
	partProps->radius = 16e-3f;
	partProps->mass = 1e-2;
	partProps->collideStiffness = 1e3;
	partProps->collideDamping = 0.2f;
	partProps->boundaryDamping = BOUNDARYDAMPING;
	
	// tamanho do quadrado que contem a esfera em PIXEL (para a saida grafica)
	sisProps->dimx = ceil(sisProps->imageDIMx/sisProps->cubeDimension.x*partProps->radius)*2;
	if (sisProps->dimx < 2) sisProps->dimx = 2;
	sisProps->dimy = ceil(sisProps->imageDIMy/sisProps->cubeDimension.y*partProps->radius)*2;
	if (sisProps->dimy < 2) sisProps->dimy = 2;
	
	// raio da esfera em PIXEL (para a saída grafica)
	sisProps->pRadius = sisProps->imageDIMy/sisProps->cubeDimension.y*partProps->radius;

	// Bloco inicial de esferas
	float corner1[2] = {0.1, 0.1}; 				 // canto inferior esquerdo
	float corner2[2] = {9.9, 9.9}; 				  // canto superior direito
	float sideLenght[2];
	sideLenght[0] = corner2[0] - corner1[0]; 			   // dimensao em X
	sideLenght[1] = corner2[1] - corner1[1]; 			   // dimensao em Y
	
	uint side[2] = {X_PARTICLES, Y_PARTICLES}; // numero de partículas em X
											  // e Y (deve ser maior que 2)

	// Calcula o tamanho do grid arredondando para um valor que seja
	// potencia de 2. O grid deve ser de 1.2 a 3 vezes o diametro da esfera
	uint grid = sisProps->cubeDimension.x / (4.0f * partProps[0].radius);
	uint temp = log2(grid);
	uint gridUpdate = 1 << temp;
	float cellSize = sisProps->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * partProps[0].radius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * partProps[0].radius ) temp += 1;
	sisProps->gridSize.x = 1 << temp;
	
	grid = sisProps->cubeDimension.y / (4 * partProps[0].radius);
	temp = log2(grid);
	gridUpdate = 1 << temp;
	cellSize = sisProps->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * partProps[0].radius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * partProps[0].radius ) temp += 1;	
	sisProps->gridSize.y = 1 << temp;

	sisProps->numCells = sisProps->gridSize.x * sisProps->gridSize.y;
	
	// alocando vetores na placa de video
	// hipMalloc --> aloca espaço na placa de vídeo
	// hipMemcpy --> transfere dados entre a CPU (Host) e GPU (Device)
	// hipMemcpyToSymbol --> copia variável para a memória de constante
	float *d_corner1, *d_sideLenght;
	uint *d_side;

	hipMalloc((void**)&d_corner1, sizeof(float)*2);
	hipMalloc((void**)&d_sideLenght, sizeof(float)*2);
	hipMalloc((void**)&d_side, sizeof(uint)*2);
	hipMalloc((void**)&partValues->pos1, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->pos2, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->vel1, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->vel2, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->acc, sizeof(float) * sisProps->numParticles * 2);
	hipMalloc((void**)&partValues->cellStart, sizeof(uint) * sisProps->numCells);
	hipMalloc((void**)&partValues->cellEnd, sizeof(uint) * sisProps->numCells);
	hipMalloc((void**)&partValues->gridParticleIndex, sizeof(uint) * sisProps->numParticles);
	hipMalloc((void**)&partValues->gridParticleHash, sizeof(uint) * sisProps->numParticles);
	hipMemcpy(d_corner1, corner1, sizeof(float)*2, hipMemcpyHostToDevice);
	hipMemcpy(d_sideLenght, sideLenght, sizeof(float)*2, hipMemcpyHostToDevice);
	hipMemcpy(d_side, side, sizeof(uint)*2, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(sisPropD), sisProps, sizeof(SystemProperties));
	hipMemcpyToSymbol(HIP_SYMBOL(partPropD), partProps , sizeof(ParticleProperties) * 1);

	// Função para definir a posição inicial das esferas
	initializeParticlePosition(partValues->pos1,
							   partValues->vel1,
							   partValues->acc,
							   d_corner1,
							   d_sideLenght,
							   d_side,
							   side,
							   time(NULL));
	
	// Desalocando espaço na placa de vídeo (Não mais necessário)
    hipFree( d_corner1 );
    hipFree( d_sideLenght );
    hipFree( d_side );

	// Screen output	
	printf("\nNumero de Particulas = %d\n",sisProps->numParticles);
	printf("grid %d x %d\n\n",sisProps->gridSize.x,sisProps->gridSize.y);
#ifdef USE_TEX
	printf("Memoria de textura: UTILIZADA\n\n");
#else
	printf("Memoria de textura: NAO\n\n");
#endif 
}

void SimLooping( uchar4 *image, DataBlock *simBlock, int ticks ) {

	// inicia o cronometro
	simBlock->start = clock();

	// Estruturas auxiliares
    SystemProperties *sisProps = &simBlock->sisProps;
    ParticlesValues *partValues = &simBlock->partValues;
	
	// para ordenarmos os vetores de posicao e velocidade sem necessidade
	// de retornarmos a variável para o vetor original, um switch entre os
	// dois vetores de posição alocados na GPU é criado. A cada iteração o
	// vetor de início e o vetor reorganizado são invertidos, reduzindo uma
	// operação de cópia
	float *oldPos, *oldVel, *sortPos, *sortVel;
	
	// Integrando o programa IPS vezes antes de exibir a imagem
	for (int i = 0 ; i < simBlock->IPS ; i++) {

		if ((ticks + i) & 1) // quando par (FALSE) quando impar (TRUE)
		{	
			oldPos = partValues->pos1;
			sortPos = partValues->pos2;
			oldVel = partValues->vel1;
			sortVel = partValues->vel2;
		} else {
			oldPos = partValues->pos2;
			sortPos = partValues->pos1;
			oldVel = partValues->vel2;
			sortVel = partValues->vel1;
		}
		
		// Integracao no tempo (atualizacao das posicoes e velocidades)
		integrateSystem(oldPos,
			 	  		oldVel,
			 	  		partValues->acc,
			 	  		sisProps->numParticles);

		// Define a celula de cada particula, criando os vetores
		// gridParticleIndex e gridParticleHash ordenados pelo Index
		calcHash(oldPos,
				 partValues->gridParticleIndex,
				 partValues->gridParticleHash,
				 sisProps->numParticles);

		// Reordena os vetores baseado no Hash
		sortParticles(partValues->gridParticleHash,
					  partValues->gridParticleIndex,
					  sisProps->numParticles);

		// Reorganiza as variaveis de Pos e Vel para a nova ordem de particulas
		// e cria os vetores indicando a partícula de início e fim de cada
		// celula
		reorderDataAndFindCellStart(partValues->cellStart,
									partValues->cellEnd,
									sortPos,
									sortVel,
									partValues->gridParticleHash,
									partValues->gridParticleIndex,
									oldPos,
									oldVel,
									sisProps->numParticles,
									sisProps->numCells);

		// Detecta a colisao das particulas e transforma a força de colisão em
		// aceleração
		collide(sortPos,
				sortVel,
				partValues->acc,
				partValues->cellStart,
				partValues->cellEnd,
				sisProps->numParticles,
				sisProps->numCells);

//		// Integracao no tempo (atualizacao das posicoes e velocidades)
//		integrateSystem(sortPos,
//			 	  		sortVel,
//			 	  		partValues->acc,
//			 	  		sisProps->numParticles);

		simBlock->tempo++;
	}

	// Saida grarica quando necessario
	plotParticles(image,
				  sortPos,
				  sisProps->numParticles,
				  sisProps->imageDIMx,
				  sisProps->imageDIMy);

	
	// calcula o tempo de exibição do frame
	double time = ((double)clock() - simBlock->start)/CLOCKS_PER_SEC;
	if (time < 0.003f) time = 0.03f;
	
	// Define o número de Interações por segundo para exibir a imagem em 
	// FPS (definida no cabeçalho) frames por segundo.
	// Após a conta, transforma o número em impar para não calcular duas
	// duas vezes a mesma iteração (por causa do switch)
	simBlock->IPS = floor(1.0f/time/FPS*simBlock->IPS);
	simBlock->IPS = simBlock->IPS | 0x0001;

}

void FinalizingSim( DataBlock *simBlock) {

    // Limpe aqui o que tiver que ser limpo
    hipFree( simBlock->partValues.pos1 );
    hipFree( simBlock->partValues.pos2 );
    hipFree( simBlock->partValues.vel1 );
    hipFree( simBlock->partValues.vel2 );
    hipFree( simBlock->partValues.acc );
    hipFree( simBlock->partValues.cellStart );
    hipFree( simBlock->partValues.cellEnd );
    hipFree( simBlock->partValues.gridParticleIndex );
    hipFree( simBlock->partValues.gridParticleHash );
    
   	printf("Integracoes por plot = %d\n\n",simBlock->IPS);
	double elapsedTime = ((double)clock() - simBlock->totalStart)/CLOCKS_PER_SEC;
	double simulationTime = simBlock->tempo * simBlock->sisProps.timeStep;
	
	printf("Duracao da simulacao = %4.2f s\n",elapsedTime);
	printf("Tempo de simulacao = %4.2f s\n\n",simulationTime);
	printf("Razao de tempo (Real/Simulado) = %3.3f\n\n",elapsedTime/simulationTime);
	
}


int main() {
	
	// declarando estrutura de dados principal
    DataBlock simBlock;
    
    // declarando as subestruturas (apenas por facilidade)
    SystemProperties *sisProps = &simBlock.sisProps;
    ParticleProperties *partProps = &simBlock.partProps;
    ParticlesValues *partValues = &simBlock.partValues;
    
    // Definindo que a primeira iteração será exibida
    simBlock.IPS = 1;
	simBlock.totalStart = clock();
	simBlock.tempo = 0;
    
    // função que define o tamanho da imagem e a estrutura que será
    // repassada para dentro do looping
    GPUAnimBitmap bitmap(DIM, DIM, &simBlock );

	// Utilizar ARGC e ARGV para pegar propriedades na linha de comando
	// ler esses comandos de um arquivo TXT externo
	// Criar uma rotina para fazer este tipo de leitura
	
	// Prepara a simulacao, define as condicoes iniciais do problema
	PrepareSim(sisProps, partValues, partProps);
	
	// Executa o looping até que a tecla ESC seja pressionada
    bitmap.anim_and_exit(
        (void (*)(uchar4*,void*,int))SimLooping, (void (*)(void*))FinalizingSim );

}
