#include "hip/hip_runtime.h"
/*
 *   DemSo - 2D Discrete Element Method for Soil application
 *   Copyright (C) 2012  UNICAMP FEM/DMC
 *
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

// Input and Output include
#include <iostream>								  // entra e saída de dados
#include <ctime> 		   // biblioteca de tempo para criar o seed do rand

// CUDA includes
#include "gpu_anim.h" 				  // bib. de vizualização em tempo real
#include "cutil_math.h" 		      // funções matemáticas de vetores

// Dependece files
#include "main.cuh"
#include "functions.cuh" 	 // arquivo de funções de preparação para a GPU
#include "datatypes.hpp"
#include "parser.hpp"


#define log2( x ) log(x)/log(2)

using std::cout;
using std::endl;

bool file_exist(const char *filename){
	
	if (FILE *file = fopen(filename,"r")){
		fclose(file);
		return true;
	}else{
		return false;
	}
}

void PrepareSim( const char *filename,
				 SystemProperties *sisProps,
				 ParticlesValues *partValues,
				 ParticleProperties *partProps,
				 RenderParameters *renderPar ) {

	/* Usamos a estrutura de dados C++ e carregamos o arquivo de estado */
	DEMSimulation sim;
	sim.loadFromFile(filename);
//	sim.printConfiguration();
	/* Agora vamos copiar para a estrutura C */

	sisProps->numParticles = sim.particles.num.x * sim.particles.num.y + 1; // Esse 1 é devido a partícula controlada ************************************************************

	sisProps->cubeDimension = make_float2(sim.environment.dimension);
	
	sisProps->timeStep = sim.parameters.timeStep;
	
	sisProps->gravity = make_float2(sim.environment.gravity); // Transformando a gravidade de float3 para float2
	
	renderPar->imageDIMx = DIM; //TODO: Fazer uma funcão q pega o ratio do environment e aplica nos imageDIM
	renderPar->imageDIMy = DIM;

	// PARSER: copiando as propriedades de partículas
	for (register int i = 0; i < sim.properties.particleTypes.size(); i++)
	{
		partProps[i].mass = sim.properties.particleTypes[i].mass;
		partProps[i].radius = sim.properties.particleTypes[i].radius;
		partProps[i].collideStiffness = sim.properties.particleTypes[i].normalStiffness;
		partProps[i].collideDamping = sim.properties.particleTypes[i].normalDamping;
		partProps[i].boundaryDamping = sim.properties.particleTypes[i].boundaryDamping;
		partProps[i].colorR = sim.properties.particleTypes[i].color.x;
		partProps[i].colorG = sim.properties.particleTypes[i].color.y;
		partProps[i].colorB = sim.properties.particleTypes[i].color.z;
	}

	// Definindo o maior raio da simulação
	// Se existir uma única partícula gigante (TUBULAÇÃO) talvez seja
	// interessante desprezar esse valor e no lugar, fazer com que essa
	// única partícula teste com todas as outras.
	float maxRadius = 0;
	for (int i = 0; i < sim.properties.particleTypes.size(); i++){
		if (maxRadius < partProps[i].radius) maxRadius = partProps[i].radius;
	}
	
	// tamanho do quadrado que contem a esfera em PIXEL (para a saida grafica)
	renderPar->dimx = ceil(renderPar->imageDIMx/sisProps->cubeDimension.x*maxRadius)*2;
	if (renderPar->dimx < 2) renderPar->dimx = 2;
	renderPar->dimy = ceil(renderPar->imageDIMy/sisProps->cubeDimension.y*maxRadius)*2;
	if (renderPar->dimy < 2) renderPar->dimy = 2;
	
	// raio da esfera em PIXEL (para a saída grafica)
	renderPar->pRadius = renderPar->imageDIMy/sisProps->cubeDimension.y*maxRadius;

	// Calcula o tamanho do grid arredondando para um valor que seja
	// potencia de 2. O grid deve ser de 1.2 a 3 vezes o diametro da esfera
	uint grid = sisProps->cubeDimension.x / (4.0f * maxRadius);
	uint temp = log2(grid);
	uint gridUpdate = 1 << temp;
	float cellSize = sisProps->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * maxRadius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * maxRadius ) temp += 1;
	sisProps->gridSize.x = 1 << temp;
	
	grid = sisProps->cubeDimension.y / (4 * maxRadius);
	temp = log2(grid);
	gridUpdate = 1 << temp;
	cellSize = sisProps->cubeDimension.x / gridUpdate;
	if ( cellSize/2.0f <= 1.2f * maxRadius ) temp -= 1;
	else if (cellSize/2.0f >= 3.0f * maxRadius ) temp += 1;	
	sisProps->gridSize.y = 1 << temp;

	sisProps->numCells = sisProps->gridSize.x * sisProps->gridSize.y;
	
	// Bloco inicial de esferas
	float sideLenght[2];
	sideLenght[0] = sim.particles.end[0] - sim.particles.start[0]; 			   // dimensao em X
	sideLenght[1] = sim.particles.end[1] - sim.particles.start[1]; 			   // dimensao em Y
	
	uint side[2];
	side[0] = sim.particles.num.x;
	side[1] = sim.particles.num.y;
	
	allocateVectors(partProps, partValues, sisProps, renderPar);

	// Função para definir a posição inicial das esferas
	initializeParticlePosition(partValues->pos1,
							   partValues->vel1,
							   partValues->acc,
							   partValues->ID1,
							   partValues->loc1,
							   partValues->type1,
							   sim.particles.start,
							   sideLenght,
							   side,
							   time(NULL),
							   sim.properties.particleTypes.size()-1); // subraindo 1 para não fazer o sorteio com a partícula controlada ********************************************

	float2 bigParticlePos = make_float2(5,9.5);
	float2 bigParticleVel = make_float2(0,0);

	// Adicionar partícula externa gigante
	initializeBigParticlePosition(partValues->pos1,
								  partValues->vel1,
								  partValues->acc,
								  partValues->ID1,
								  partValues->loc1,
								  partValues->type1,
								  bigParticlePos,
								  bigParticleVel,
								  sim.properties.particleTypes.size()-1); // Tipo da partícula, por enquanto ela é a última **************************************************************
	
	// Screen output	
	printf("Numero de Particulas = %d\n",sisProps->numParticles);
	printf("grid %d x %d\n\n",sisProps->gridSize.x,sisProps->gridSize.y);
#if USE_TEX
	printf("Memoria de textura: UTILIZADA\n\n");
#else
	printf("Memoria de textura: NAO\n\n");
#endif 
}

void SimLooping( uchar4 *image, DataBlock *simBlock, int ticks ) {

	// Estruturas auxiliares
    SystemProperties *sisProps = &simBlock->sisProps;
    ParticlesValues *partValues = &simBlock->partValues;
	RenderParameters *renderPar = &simBlock->renderPar;
	TimeControl *timeCtrl = &simBlock->timeCtrl;

	// inicia o cronometro
	timeCtrl->start = clock();
	
	// para ordenarmos os vetores de posicao e velocidade sem necessidade
	// de retornarmos a variável para o vetor original, um switch entre os
	// dois vetores de posição alocados na GPU é criado. A cada iteração o
	// vetor de início e o vetor reorganizado são invertidos, reduzindo uma
	// operação de cópia
	float  *oldPos,  *oldVel;
	float *sortPos, *sortVel;
	uint  *oldID,  *oldType,  *oldLoc;
	uint *sortID, *sortType, *sortLoc;
	
	// Integrando o programa IPS vezes antes de exibir a imagem
	for (int i = 0 ; i < timeCtrl->IPS ; i++) {

		if ((ticks + i) & 1) // quando par (FALSE) quando impar (TRUE)
		{	
			oldPos = partValues->pos1;
			oldVel = partValues->vel1;
			oldID = partValues->ID1;
			oldLoc = partValues->loc1;
			oldType = partValues->type1;
			sortPos = partValues->pos2;
			sortVel = partValues->vel2;
			sortID = partValues->ID2;
			sortLoc = partValues->loc2;
			sortType = partValues->type2;
		} else {
			oldPos = partValues->pos2;
			oldVel = partValues->vel2;
			oldID = partValues->ID2;
			oldLoc = partValues->loc2;
			oldType = partValues->type2;
			sortPos = partValues->pos1;
			sortVel = partValues->vel1;
			sortID = partValues->ID1;
			sortLoc = partValues->loc1;
			sortType = partValues->type1;
		}
		
//		// Integracao no tempo (atualizacao das posicoes e velocidades)
//		integrateSystem(oldPos,
//			 	  		oldVel,
//			 	  		partValues->acc,
//			 	  		oldType,
//			 	  		sisProps->numParticles);

		// Define a celula de cada particula, criando os vetores
		// gridParticleIndex e gridParticleHash ordenados pelo Index
		calcHash(oldPos,
				 partValues->gridParticleIndex,
				 partValues->gridParticleHash,
				 sisProps->numParticles);

		// Reordena os vetores baseado no Hash
		sortParticles(partValues->gridParticleHash,
					  partValues->gridParticleIndex,
					  sisProps->numParticles);

		// Reorganiza as variaveis de Pos e Vel para a nova ordem de particulas
		// e cria os vetores indicando a partícula de início e fim de cada
		// celula
		reorderDataAndFindCellStart(partValues->cellStart,
									partValues->cellEnd,
									sortPos,
									sortVel,
									sortID,
									sortLoc,
									sortType,
									partValues->gridParticleHash,
									partValues->gridParticleIndex,
									oldPos,
									oldVel,
									oldID,
									oldType,
									sisProps->numParticles,
									sisProps->numCells);

		// Detecta a colisao das particulas e transforma a força de colisão em
		// aceleração
		collide(sortPos,
				sortVel,
				partValues->acc,
				sortType,
				partValues->cellStart,
				partValues->cellEnd,
				sisProps->numParticles,
				sisProps->numCells);

		// Integracao no tempo (atualizacao das posicoes e velocidades)
		integrateSystem(sortPos,
			 	  		sortVel,
			 	  		partValues->acc,
						sortType,
			 	  		sisProps->numParticles);
			 	  		
		restoreFixPositions(oldPos,
							sortPos,
							oldLoc,
							sortLoc);

		timeCtrl->tempo++;
	}

	// Saida grarica quando necessario
	plotParticles(image,
				  sortPos,
				  sortType,
				  sisProps->numParticles,
				  renderPar->imageDIMx,
				  renderPar->imageDIMy);

	
	// calcula o tempo de exibição do frame
	double time = ((double)clock() - timeCtrl->start)/CLOCKS_PER_SEC;
	if (time < 0.003f) time = 0.03f;
	
	// Define o número de Interações por segundo para exibir a imagem em 
	// FPS (definida no cabeçalho) frames por segundo.
	// Após a conta, transforma o número em impar para não calcular duas
	// duas vezes a mesma iteração (por causa do switch)
	timeCtrl->IPS = floor(1.0f/time/FPS*timeCtrl->IPS);
	timeCtrl->IPS = timeCtrl->IPS | 0x0001;

}

void FinalizingSim( DataBlock *simBlock) {

	TimeControl *timeCtrl = &simBlock->timeCtrl;

    // Limpe aqui o que tiver que ser limpo
	desAllocateVectors( &simBlock->partValues );
    
   	printf("Integracoes por plot = %d\n\n",timeCtrl->IPS);
	double elapsedTime = ((double)clock() - timeCtrl->totalStart)/CLOCKS_PER_SEC;
	double simulationTime = timeCtrl->tempo * simBlock->sisProps.timeStep;
	
	printf("Duracao da simulacao = %4.2f s\n",elapsedTime);
	printf("Tempo de simulacao = %4.2f s\n\n",simulationTime);
	printf("Razao de tempo (Real/Simulado) = %3.3f\n\n",elapsedTime/simulationTime);
	
}


int main(int argc, char **argv) {
	
	// Verificando arquivo de entrada (Parametros da simulacao)
	const char *filename;
	
	if (argc == 2){
		if (file_exist(argv[1])){
			printf("\nUsing %s parameters file\n\n",argv[1]);
			filename = argv[1];
		}else if (file_exist("exemplos/default.dsml")){
			printf("\nFile %s does not exist, using exemplos/default.dsml file\n\n",argv[1]);
			filename = "exemplos/default.dsml";
		}else{
			printf("\nFile %s and exemlos/default.dsml does not exist.\n\nClosing simulation\n\n",argv[1]);
			return 0;
		}
	}else if (argc == 1){
		if (file_exist("exemplos/default.dsml")){
			printf("\nUsing default parameters file (exemplos/default.dsml)\n\n");
			filename = "exemplos/default.dsml";
		}else{
			printf("\nDefault file exemlos/default.dsml does not exist.\n\nClosing simulation\n\n");
			return 0;
		}
	}else{
		printf("\nToo many arguments.\n\nClosing simulation\n\n");
		return 0;
	}
	
	// declarando estrutura de dados principal
    DataBlock simBlock;
    
    // declarando as subestruturas (apenas por facilidade)
    SystemProperties *sisProps = &simBlock.sisProps;
    ParticlesValues *partValues = &simBlock.partValues;
	RenderParameters *renderPar = &simBlock.renderPar;
	TimeControl *timeCtrl = &simBlock.timeCtrl;
    
    ParticleProperties partProps[MAX_PARTICLES_TYPES];
    
    // Definindo que a primeira iteração será exibida
    timeCtrl->IPS = 1;
	timeCtrl->totalStart = clock();
	timeCtrl->tempo = 0;
    
    // função que define o tamanho da imagem e a estrutura que será
    // repassada para dentro do looping
    GPUAnimBitmap bitmap(DIM, DIM, &simBlock );
	
	// Prepara a simulacao, define as condicoes iniciais do problema
	PrepareSim(filename, sisProps, partValues, partProps, renderPar);
	
	// Executa o looping até que a tecla ESC seja pressionada
    bitmap.anim_and_exit(
        (void (*)(uchar4*,void*,int))SimLooping, (void (*)(void*))FinalizingSim );

}
